#include "hip/hip_runtime.h"
#include "test_cuda.h"
#include "CudaManager.h"
#include "SourceFFT_Cuda.h"
#include "LocalCodesFFT_Cuda.h"
#include "ContinuousPhaseCarrier.h"
#include "GoldCodeGenerator.h"
#include "CodeModulator_BPSK.h"
#include "Cuda_Operators.h"
#include "Cuda_IndexTransforms.h"
#include "WsgcException.h"

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include <thrust/transform.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <hipfft/hipfft.h>
#include <hipblas.h>
#include <cutil_inline.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h


test_cuda::test_cuda(options_t& options) :
	_options(options)
{
	_cuda_manager = new CudaManager(options.nb_message_symbols, options.nb_pilot_prns);
    _cuda_manager->diagnose();
    std::ostringstream cuda_os;
    _cuda_manager->dump(cuda_os);
    std::cout << cuda_os.str() << std::endl << std::endl;
}


test_cuda::~test_cuda()
{
	delete _cuda_manager;
}


void test_cuda::test1()
{
    ContinuousPhaseCarrier source_oscillator(_options.f_sampling, _options.fft_N);
    source_oscillator.make_next_samples(_options.f_chip);

    SourceFFT_Cuda source_fft(_options.f_sampling, _options.f_chip, _options.fft_N, _options.freq_step_division);

    thrust::device_vector<hipComplex>& d_fft_source = source_fft.do_fft(source_oscillator.get_samples());
    thrust::host_vector<hipComplex> h_fft_source(_options.freq_step_division*_options.fft_N);
    thrust::copy(d_fft_source.begin(), d_fft_source.end(), h_fft_source.begin());

    thrust::host_vector<hipComplex>::iterator it = h_fft_source.begin();
    const thrust::host_vector<hipComplex>::iterator it_end = h_fft_source.end();
    unsigned int i = 0;

    for (;it != it_end; ++it, i++)
    {
    	wsgc_float module = (*it).x * (*it).x + (*it).y * (*it).y;
    	std::cout << i/_options.fft_N << "," << i % _options.fft_N << ": " << module << std::endl;
    }
}


void test_cuda::test2(wsgc_complex *message_samples, GoldCodeGenerator& gc_generator, CodeModulator_BPSK& code_modulator)
{
	std::cout << "--- input:" << std::endl;

	for (unsigned int i=0; i < _options.fft_N; i++)
	{
		std::cout << i << ": (" << message_samples[i].real() << "," << message_samples[i].imag() << ")" << std::endl;
	}

	SourceFFT_Cuda source_fft(_options.f_sampling, _options.f_chip, _options.fft_N, _options.freq_step_division);
	thrust::device_vector<hipComplex>& d_fft_source = source_fft.do_fft(message_samples);

	LocalCodesFFT_Cuda local_codes(code_modulator, gc_generator, _options.f_sampling, _options.f_chip, _options.prn_list);

	thrust::device_vector<hipComplex> d_result(_options.fft_N);
	const thrust::device_vector<hipComplex>& d_local_codes = local_codes.get_local_codes();

	std::cout << d_fft_source.size() << ":" << d_local_codes.size() << std::endl;

	thrust::transform(d_fft_source.begin(), d_fft_source.begin()+_options.fft_N, d_local_codes.begin(), d_result.begin(), cmulc_functor2());

	hipfftHandle ifft_plan;
	hipfftResult_t fft_stat = hipfftPlan1d(&ifft_plan, _options.fft_N, HIPFFT_C2C, 1);
	thrust::device_vector<hipComplex> d_ifft(_options.fft_N);

    if (hipfftExecC2C(ifft_plan, thrust::raw_pointer_cast(&d_result[0]), thrust::raw_pointer_cast(&d_ifft[0]), HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
    {
        throw WsgcException("CUFFT Error: Failed to do IFFT of source*local_code");
    }

    thrust::host_vector<hipComplex> h_ifft(_options.fft_N);
    thrust::host_vector<hipComplex>::iterator it = h_ifft.begin();
    const thrust::host_vector<hipComplex>::iterator it_end = h_ifft.end();
    unsigned int i = 0;

    thrust::copy(d_result.begin(), d_result.begin()+_options.fft_N, h_ifft.begin());

    std::cout << "--- input IFFT:" << std::endl;

    for (;it != it_end; ++it, i++)
    {
    	std::cout << i << ": (" << (*it).x << "," << (*it).y << ")" << std::endl;
    }

    thrust::copy(d_ifft.begin(), d_ifft.end(), h_ifft.begin());

    it = h_ifft.begin();
    i = 0;

    std::cout << "--- result IFFT:" << std::endl;

    for (;it != it_end; ++it, i++)
    {
    	std::cout << i << ": (" << (*it).x << "," << (*it).y << ")" << std::endl;
    }

	hipfftDestroy(ifft_plan);
}


void test_cuda::test3(wsgc_complex *message_samples, GoldCodeGenerator& gc_generator, CodeModulator_BPSK& code_modulator)
{
	SourceFFT_Cuda source_fft(_options.f_sampling, _options.f_chip, _options.fft_N, _options.freq_step_division);
	thrust::device_vector<hipComplex>& d_source_block = source_fft.do_fft(message_samples);

	LocalCodesFFT_Cuda local_codes(code_modulator, gc_generator, _options.f_sampling, _options.f_chip, _options.prn_list);
	const thrust::device_vector<hipComplex>& d_local_codes = local_codes.get_local_codes();

	thrust::device_vector<hipComplex> d_ifft_in(_options.fft_N*_options.nb_f_bins*_options.freq_step_division*2*_options.nb_batch_prns);
	thrust::device_vector<hipComplex> d_ifft_out(_options.fft_N*_options.nb_f_bins*_options.freq_step_division*2*_options.nb_batch_prns);
	thrust::host_vector<hipComplex> h_ifft_out(_options.fft_N*_options.nb_f_bins*_options.freq_step_division*2*_options.nb_batch_prns);

	unsigned int pilot_prn_index = 0;
	unsigned int prn_position = 0;

    thrust::for_each(
        thrust::make_zip_iterator(
            thrust::make_tuple(
                thrust::make_permutation_iterator(d_source_block.begin(), thrust::make_transform_iterator(thrust::make_counting_iterator(0), transpose_index_A(_options.fft_N, _options.freq_step_division))),
                thrust::make_permutation_iterator(d_local_codes.begin() + pilot_prn_index*_options.fft_N, thrust::make_transform_iterator(thrust::make_counting_iterator(0), transpose_index_B(_options.fft_N, _options.freq_step_division, _options.nb_f_bins))),
                thrust::make_permutation_iterator(d_ifft_in.begin(), thrust::make_transform_iterator(thrust::make_counting_iterator(0), transpose_index_C(_options.nb_batch_prns, prn_position)))
            )
        ),
        thrust::make_zip_iterator(
            thrust::make_tuple(
				thrust::make_permutation_iterator(d_source_block.begin(), thrust::make_transform_iterator(thrust::make_counting_iterator(0)+(_options.fft_N*_options.freq_step_division*_options.nb_f_bins), transpose_index_A(_options.fft_N, _options.freq_step_division))),
				thrust::make_permutation_iterator(d_local_codes.begin() + (pilot_prn_index+1)*_options.fft_N, thrust::make_transform_iterator(thrust::make_counting_iterator(0)+(_options.fft_N*_options.freq_step_division*_options.nb_f_bins), transpose_index_B(_options.fft_N, _options.freq_step_division, _options.nb_f_bins))),
				thrust::make_permutation_iterator(d_ifft_in.begin(), thrust::make_transform_iterator(thrust::make_counting_iterator(0)+(_options.fft_N*_options.freq_step_division*_options.nb_f_bins), transpose_index_C(_options.nb_batch_prns, prn_position)))
            )
        ),
        cmulc_functor()
    );

    hipfftHandle ifft_plan;

	int n[1];                                       //!< CUFFT Plan FFT size parameter
	int inembed[1];                                 //!< CUFFT Plan parameter
	int onembed[1];                                 //!< CUFFT Plan parameter

    n[0] = _options.fft_N;
    inembed[0] = _options.fft_N;
    onembed[0] = _options.fft_N;

    hipfftResult_t fft_stat = hipfftPlanMany(&ifft_plan, 1, n,
		inembed, 2*_options.nb_batch_prns, 2*_options.nb_batch_prns*_options.fft_N,
		onembed, 2*_options.nb_batch_prns, 2*_options.nb_batch_prns*_options.fft_N,
		HIPFFT_C2C, _options.nb_f_bins*_options.freq_step_division);

    if (fft_stat != HIPFFT_SUCCESS)
    {
    	std::ostringstream err_os;
    	err_os << "CUFFT Error: Unable to create plan for pilot IFFT RC=" << fft_stat;
    	throw WsgcException(err_os.str());
    }

    if (hipfftExecC2C(ifft_plan,
    		         thrust::raw_pointer_cast(&d_ifft_in[prn_position]),
    		         thrust::raw_pointer_cast(&d_ifft_out[prn_position]),
    		         HIPFFT_BACKWARD) != HIPFFT_SUCCESS)
    {
    	throw WsgcException("CUFFT Error: Failed to execute IFFT");
    }

    thrust::copy(d_ifft_out.begin(), d_ifft_out.end(), h_ifft_out.begin());

    std::cout << "--- f0.0 :" << std::endl;

    unsigned int f_shift = 0;

    /*
    for (unsigned int i=0; i < _options.fft_N; i++)
    {
    	unsigned int mi = f_shift + prn_position + i*2*_options.nb_batch_prns;
    	std::cout << i << ">" << mi << ": (" << h_ifft_out[mi].x << "," << h_ifft_out[mi].y << ")" << std::endl;
    }
    */

    std::cout << "--- f1.0 :" << std::endl;

    f_shift = _options.fft_N * 2 *_options.nb_batch_prns * _options.freq_step_division;

    for (unsigned int i=0; i < _options.fft_N; i++)
    {
    	unsigned int mi = f_shift + prn_position + i*2*_options.nb_batch_prns;
    	std::cout << i << ">" << mi << ": (" << h_ifft_out[mi].x << "," << h_ifft_out[mi].y << ")" << std::endl;
    }

    std::cout << "--- f1.1 :" << std::endl;

    f_shift = _options.fft_N * 2 *_options.nb_batch_prns * (_options.freq_step_division+1);

    for (unsigned int i=0; i < _options.fft_N; i++)
    {
    	unsigned int mi = f_shift + prn_position + i*2*_options.nb_batch_prns;
    	std::cout << i << ">" << mi << ": (" << h_ifft_out[mi].x << "," << h_ifft_out[mi].y << ")" << std::endl;
    }

    hipblasHandle_t cublas_handle;

	hipblasStatus_t stat = hipblasCreate(&cublas_handle);
	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		throw WsgcException("CUBLAS Error: Failed to initialize library");
	}

    int cublas_max_index;

	stat = hipblasIcamax(cublas_handle, (_options.fft_N*_options.freq_step_division*_options.nb_f_bins),
			thrust::raw_pointer_cast(&d_ifft_out[prn_position]),
			2*_options.nb_batch_prns, &cublas_max_index);

	if (stat != HIPBLAS_STATUS_SUCCESS)
	{
		std::ostringstream err_os;
		err_os << "CUBLAS Error: hipblasIcamax failed with RC=" << stat;
		std::cout << err_os.str() << std::endl;
		throw WsgcException(err_os.str());
	}

	cublas_max_index--; // Cublas index are 1 based

    hipComplex z = d_ifft_out[prn_position + cublas_max_index*2*_options.nb_batch_prns];
    std::cout << prn_position << ": IFFT max: " << prn_position + cublas_max_index*2*_options.nb_batch_prns << " : " << mag_algebraic_functor()(z) << std::endl;
    std::cout << cublas_max_index % _options.fft_N << std::endl;
    std::cout << cublas_max_index / _options.fft_N << std::endl;

    hipfftDestroy(ifft_plan);
    hipblasDestroy(cublas_handle);
}
