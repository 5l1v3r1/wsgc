#include "Cuda_Operators.h"

std::ostream& operator<<(std::ostream& os, const hipComplex& z)
{
    os << "(" << z.x << " , " << z.y << ")";
    return os;
}