/*
     Copyright 2012 Edouard Griffiths <f4exb at free dot fr>
 
     This file is part of WSGC. A Weak Signal transmission mode using Gold Codes
 
     This program is free software; you can redistribute it and/or modify
     it under the terms of the GNU General Public License as published by
     the Free Software Foundation; either version 2 of the License, or
     (at your option) any later version.
 
     This program is distributed in the hope that it will be useful,
     but WITHOUT ANY WARRANTY; without even the implied warranty of
     MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
     GNU General Public License for more details.
 
     You should have received a copy of the GNU General Public License
     along with this program; if not, write to the Free Software
     Foundation, Inc., 51 Franklin Street, Boston, MA  02110-1301  USA

     Static not real time prototype in C++
      
     LocalCodes - CUDA implementation
      
     Creates a local copy of the complex conjugate of the FFT transform of all symbols gold codes
     This pre-calculates the code to be inserted in the final IFFT.
     Used with BPSK complex signals.
     
*/
#include "LocalCodesFFT_Cuda.h"
#include "GoldCodeGenerator.h"
#include "CodeModulator.h"
#include "WsgcException.h"
#include "Cuda_Operators.h"
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <hip/hip_complex.h>
#include <cutil_inline.h>    // includes hip/hip_runtime.h and hip/hip_runtime_api.h
#include <hipfft/hipfft.h>
#include <string.h>
#include <assert.h>

LocalCodesFFT_Cuda::LocalCodesFFT_Cuda(CodeModulator& code_modulator, 
            const GoldCodeGenerator& gc_generator,
            wsgc_float f_sampling, 
            wsgc_float f_chip,
            std::vector<unsigned int>& symbols,
            unsigned int cuda_device) :
    CudaDeviceManager::CudaDeviceManager(cuda_device),
    LocalCodesFFT(code_modulator, gc_generator, f_sampling, f_chip, symbols),
    _nb_codes(symbols.size()),
    _d_code(_nb_code_samples*symbols.size())
{
    _h_fft_code_in = new wsgc_complex[_nb_code_samples];
    hipfftResult_t fft_stat = hipfftPlan1d(&_fft_plan, _nb_code_samples, HIPFFT_C2C, _nb_codes);

    if (fft_stat != HIPFFT_SUCCESS)
    {
    	std::ostringstream os;
    	os << "CUFFT error: Failed to allocate FFT plan rc=" << fft_stat;
    	throw WsgcException(os.str());
    }

    fill_codes_matrix();
}


LocalCodesFFT_Cuda::~LocalCodesFFT_Cuda()
{
    hipfftDestroy(_fft_plan);
    delete[] _h_fft_code_in;
}


void LocalCodesFFT_Cuda::fill_codes_matrix()
{
    std::vector<char> code;
    thrust::device_vector<hipComplex> _d_fft_code_in(_nb_codes*_nb_code_samples);
    
    // Fill input codes samples
    
    std::vector<unsigned int>::iterator prni_it = _symbols.begin();
    const std::vector<unsigned int>::iterator prni_end = _symbols.end();
    unsigned int i=0;
    
    for (; prni_it != prni_end; ++prni_it, i++)
    {
        assert(*prni_it < _gc_generator.get_nb_codes());

        _gc_generator.make_code(code, *prni_it); // 0/1 bits
        _code_modulator.fill_code_samples(reinterpret_cast<wsgc_fftw_complex *>(_h_fft_code_in), code, _f_sampling, _f_chip);   // This is the modulation specific part
        
        // copy to device vector of codes
        thrust::copy(
            reinterpret_cast<const hipComplex *>(_h_fft_code_in), 
            reinterpret_cast<const hipComplex *>(_h_fft_code_in+_nb_code_samples), 
            _d_fft_code_in.begin() + (i*_nb_codes)
        );

        index_symbol(i, *prni_it);
    }
    
    // Do the FFT
    
    if (hipfftExecC2C(_fft_plan, thrust::raw_pointer_cast(&_d_fft_code_in[0]), thrust::raw_pointer_cast(&_d_fft_code_in[0]), HIPFFT_FORWARD) != HIPFFT_SUCCESS)
    {
        throw WsgcException("CUFFT Error: Failed to do FFT of local code");
    }
        
    // Do the conjugate
    
    thrust::transform(_d_fft_code_in.begin(), _d_fft_code_in.end(), _d_code.begin(), conj_functor());
}
        
